#include "hip/hip_runtime.h"


#include <stdio.h>

// this function is run in the gpu
__global__ void vectorAdd(int* a, int* b, int* c)
{
    //we are creating a list of threads and the .x will indicate the vecotr where we are
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

    return;
}

int main()
{
    // larger values
    int a[] = { 1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3, };
    int b[] = { 4,5,6,4,5,6,4,5,6,4,5,6,4,5,6,4,5,6,4,5,6,4,5,6, };
    // original values
    //int a[] = { 1,2,3, };
    //int b[] = { 4,5,6 };
    int c[sizeof(a) / sizeof(int)] = { 0 };

    //for (int i = 0; i < sizeof(c) / sizeof(int); i++)
    //{
    //    c[i] = a[i] + b[i];
    //}

    // create pointers into the gpu
    int* cudaA = 0;
    int* cudaB = 0;
    int* cudaC = 0;
    
    // allocate memory in the gpu
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    // copy the vectors into the gpu
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
    //cudaMemcpy(cudaC, c, sizeof(a), cudaMemcpyHostToDevice);
    
    // this needs to be instantiated
    //vectorAdd<<<GRID_SIZE, BLOCK_SIZE
    // GRID_SIZE is the number of blocks that it has
    // per the amount of blocks it says how many threads exist per block
    // block size is the number of vectors or threads
    
    //explanation
    //run this function vectorAdd (<<< in a cuda kernel) in a grid that has 1 block, with the block have this many threads (>>> and call it with these parameters)
    vectorAdd <<< 1, sizeof(a) / sizeof(int) >>> (cudaA, cudaB, cudaC);

    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

    return;
}

//
// This is code just running on cpu
//
//    int a[] = { 1,2,3 };
//    int b[] = { 4,5,6 };
//    int c[sizeof(a) / sizeof(int)] = { 0 };
//    
//    for (int i = 0; i < sizeof(c) / sizeof(int); i++)
//    {
//        c[i] = a[i] + b[i];
//    }
//
//    return;
